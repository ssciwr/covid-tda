#include "hip/hip_runtime.h"
#include "cuda_mem.hh"
#include "hamming/distance_cuda.hh"
#include "hamming/hamming_utils.hh"
#include <chrono>
#include <cuda/std/limits>

namespace hamming {

template <typename DistIntType>
__global__ void Dist(DistIntType *partial_distances, const std::uint8_t *genes,
                     std::uint64_t distances_offset,
                     unsigned int geneBlocksPerSample) {
  // Calculates all gridDim.x entries of the partial_distances array.
  //
  // The full distances array is a flat nsamples * (nsamples - 1) / 2 element
  // array that contains the lower-triangular elements of the nSamples x
  // nSamples partial_distances matrix.
  //
  // This kernel is provided with partial_distances, which should have
  // gridDim.x entries, and which is filled with values corresponding to
  // entries in the full distances array with an offset of distances_offset.
  //

  // this array is shared between the threads in this block
  // and must be large enough to store one int per thread
  extern __shared__ int s[];

  // index in the partial_distances array where we'll put the result from this
  // block of threads
  uint64_t distancesIndex{static_cast<uint64_t>(blockIdx.x)};
  // index of this value in the full distances array
  uint64_t trueDistancesIndex{distancesIndex + distances_offset};
  // infer indices of the two genes corresponding to this distances index
  uint64_t distancesRowIndex{static_cast<std::size_t>(
      floor(sqrt(2.0 * static_cast<double>(trueDistancesIndex) + 0.5) + 0.5))};
  uint64_t distancesColIndex{trueDistancesIndex -
                             distancesRowIndex * (distancesRowIndex - 1) / 2};
  uint64_t uint32sPerSample{geneBlocksPerSample / 4};
  uint64_t geneAIndex{distancesRowIndex * uint32sPerSample};
  uint64_t geneBIndex{distancesColIndex * uint32sPerSample};

  unsigned int threadIndex{threadIdx.x};
  // calculate partial sum for each thread and store in shared memory s
  int r0{0};
  int r1{0};
  int r2{0};
  int r3{0};
  // NOTE: this cast is only safe if genes is 32-bit aligned AND we each sample
  // in genes is padded such that the first element of each sample is also
  // 32-bit aligned!
  const uint32_t *genes_as_uint32{reinterpret_cast<const uint32_t *>(genes)};
  uint mask_lower{0x0f0f0f0f};
  uint mask_upper{0xf0f0f0f0};
  // NOTE: this loop is also only correct if the length of each sample in genes
  // is a multiple of 8, which we do by padding the samples with '-'.
  for (int j = 2 * threadIndex; j < uint32sPerSample; j += 2 * blockDim.x) {
    auto c0{genes_as_uint32[geneAIndex + j] & genes_as_uint32[geneBIndex + j]};
    auto c1{genes_as_uint32[geneAIndex + j + 1] &
            genes_as_uint32[geneBIndex + j + 1]};
    r0 += __popc(__vseteq4(c0 & mask_lower, 0u));
    r1 += __popc(__vseteq4(c0 & mask_upper, 0u));
    r2 += __popc(__vseteq4(c1 & mask_lower, 0u));
    r3 += __popc(__vseteq4(c1 & mask_upper, 0u));
  }
  s[threadIndex] = r0 + r1 + r2 + r3;
  // synchronise shared memory s between all threads in this block
  __syncthreads();
  // sum elements of s using reduction until partial sums are stored in
  // the first 64 elements of s
  for (int offset = blockDim.x / 2; offset > 32; offset >>= 1) {
    if (threadIndex < offset) {
      s[threadIndex] += s[threadIndex + offset];
    }
    __syncthreads();
  }
  if (threadIndex < 32) {
    // one more reduction in each of the 32 threads in this warp
    int sum{s[threadIndex] + s[threadIndex + 32]};
    // now sum the values of sum within this warp
    constexpr unsigned int FULL_MASK{0xffffffff};
    for (int offset = 16; offset > 0; offset /= 2) {
      sum += __shfl_down_sync(FULL_MASK, sum, offset);
    }
    if (threadIndex == 0) {
      auto maxDist{cuda::std::numeric_limits<DistIntType>::max()};
      partial_distances[distancesIndex] = sum > maxDist ? maxDist : sum;
    }
  }
}

template <typename DistIntType>
std::vector<DistIntType>
distances_cuda(const std::vector<std::vector<GeneBlock>> &data,
               const std::string &filename = {}) {
  std::vector<DistIntType> distances{};
  std::size_t timing_gpu_ms = 0;
  std::size_t timing_io_ms = 0;
  auto timing0{std::chrono::high_resolution_clock::now()};
  bool output_to_vector{false};
  if (filename.empty()) {
    output_to_vector = true;
  }
  std::size_t nSamples{data.size()};
  std::size_t nDistances{nSamples * (nSamples - 1) / 2};
  std::size_t geneBlocksPerSample{data[0].size()};
  // 2^31-1 is limit on number of CUDA blocks in x-dim, which corresponds to
  // 0.5/1GB of distances data for each chunk. For large datasets I/O becomes
  // the bottleneck and the larger the chunk the faster the I/O tends to be.
  std::size_t nPartialDistances{std::min(nDistances, 2147483647ul)};

  if (output_to_vector) {
    // need to store all distances on host
    distances.resize(nDistances);
  } else {
    // only need to store a single block of partial distances on host
    distances.resize(nPartialDistances);
  }
  // allocate memory for genes on device
  // one gene is 30k chars -> 15k bytes in dense format
  // so 1 million samples -> 15GB
  auto *genes{CheckedCudaMalloc<GeneBlock>(nSamples * geneBlocksPerSample)};
  // copy genes to device
  for (std::size_t i = 0; i < data.size(); ++i) {
    CheckedCopyToDevice(genes + i * geneBlocksPerSample, data[i]);
  }

  // allocate memory for partial distances matrix on device
  auto *partial_distances{CheckedCudaMalloc<DistIntType>(nPartialDistances)};
  // keep track of how many distance elements are available to write to disk
  std::size_t available_distance_elements{0};
  // keep track of where in the full distances array these elements should go
  std::size_t distances_offset{0};
  // GPU timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // I/O timing
  auto timing_io = std::chrono::high_resolution_clock::now();
  while (distances_offset + available_distance_elements < nDistances) {
    // use nThreadsPerBlock in x dim of block
    uint nThreadsPerBlock{128};
    dim3 threadsPerBlock{nThreadsPerBlock, 1, 1};
    // use up to nPartialDistances blocks, one block per distance element
    dim3 numBlocks{static_cast<uint>(std::min(nPartialDistances,
                                              nDistances - distances_offset)),
                   1, 1};
    hipEventRecord(start);
    timing_io = std::chrono::high_resolution_clock::now();
    // launch a kernel with shared memory of size int[nThreadsPerBlock] -
    // this call returns immediately and the kernel runs asynchronously on the
    // GPU
    Dist<<<numBlocks, threadsPerBlock, nThreadsPerBlock * sizeof(int)>>>(
        partial_distances, genes, distances_offset, geneBlocksPerSample);
    hipEventRecord(stop);
    if (auto err = hipGetLastError(); err != hipSuccess) {
      throw std::runtime_error(hipGetErrorString(err));
    }

    if (!output_to_vector) {
      // write previous kernel's output (if any) to disk using CPU while the new
      // kernel is running on the GPU to interleave I/O with computation
      partial_write_lower_triangular(filename, distances, distances_offset,
                                     available_distance_elements);
    }
    timing_io_ms += std::chrono::duration_cast<std::chrono::milliseconds>(
                        std::chrono::high_resolution_clock::now() - timing_io)
                        .count();
    // copy partial_distances from GPU to distances vector on HOST - this call
    // waits until the kernel has completed before copying the memory.
    CheckedCopyToHost(distances.data() +
                          (output_to_vector ? distances_offset : 0),
                      partial_distances, numBlocks.x);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    timing_gpu_ms += static_cast<std::size_t>(milliseconds);
    distances_offset += available_distance_elements;
    available_distance_elements = numBlocks.x;
  }
  if (!output_to_vector) {
    // write final kernel's output to disk
    timing_io = std::chrono::high_resolution_clock::now();
    partial_write_lower_triangular(filename, distances, distances_offset,
                                   available_distance_elements);
    timing_io_ms += std::chrono::duration_cast<std::chrono::milliseconds>(
                        std::chrono::high_resolution_clock::now() - timing_io)
                        .count();
    distances.clear();
  }
  // free data on gpu
  hipFree(genes);
  hipFree(partial_distances);
  std::cout << "# hammingdist :: ...distance calculation completed in "
            << std::chrono::duration_cast<std::chrono::milliseconds>(
                   std::chrono::high_resolution_clock::now() - timing0)
                   .count()
            << " ms (GPU: " << timing_gpu_ms << " / IO: " << timing_io_ms
            << ")." << std::endl;
  return distances;
}

std::vector<uint8_t>
distances_cuda_8bit(const std::vector<std::vector<GeneBlock>> &data) {
  return distances_cuda<uint8_t>(data, {});
}

std::vector<uint16_t>
distances_cuda_16bit(const std::vector<std::vector<GeneBlock>> &data) {
  return distances_cuda<uint16_t>(data, {});
}

void distances_cuda_to_lower_triangular(
    const std::vector<std::vector<GeneBlock>> &data,
    const std::string &filename) {
  distances_cuda<uint16_t>(data, filename);
}

int distance_cuda(const std::vector<GeneBlock> &a,
                  const std::vector<GeneBlock> &b) {
  // wrapper for testing cuda kernel with existing distance API
  std::vector<std::vector<GeneBlock>> data{a, b};
  return distances_cuda<int>(data, {})[0];
}

bool distance_cuda_have_device() {
  int nDevices = 0;
  if (hipError_t err{hipGetDeviceCount(&nDevices)}; err != hipSuccess) {
    return false;
  }
  return nDevices > 0;
}

} // namespace hamming
